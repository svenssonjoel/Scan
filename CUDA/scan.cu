


#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h> 
#include <stdint.h> 


__global__ void silly_kernel(int n, int* in, int* out) {
  
  if (threadIdx.x == 0){ 
    int acc = 0; 
    
    for (int i = 0; i < n; ++i){ 
      acc += in[i]; 
      out[i] = acc; 
    }
  }
}







#define SIZE 10
int main(int argc, char **argv) {
  
  int *in = NULL; 
  int *out = NULL; 

  in = (int*)malloc(SIZE*sizeof(int)); 
  out = (int*)malloc(SIZE*sizeof(int)); 

  int *din = NULL; 
  int *dout = NULL;
  
  hipMalloc((void**)&din,SIZE*sizeof(int)); 
  hipMalloc((void**)&dout,SIZE*sizeof(int)); 

  // Generate some data 
  for (int i = 0; i < SIZE; i ++) { 
    in[i] = i+1;
  }
  
  
  hipMemcpy(din,in,SIZE*sizeof(int),hipMemcpyHostToDevice);

  silly_kernel<<<1,1,0>>>(SIZE,din,dout); 

  hipMemcpy(out,dout, SIZE*sizeof(int),hipMemcpyDeviceToHost); 

  hipFree(din);
  hipFree(dout);
  
  for (int i = 0; i < SIZE; ++i) {
    printf("%d ", out[i]);
  }
  printf("\n"); 

  
  

  return 0;
} 
