


#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h> 
#include <stdint.h> 

// Experiment with 3 part scans. 


__global__ void silly_kernel(int n, int* in, int* out) {
  
  if (threadIdx.x == 0){ 
    int acc = 0; 
    
    for (int i = 0; i < n; ++i){ 
      acc += in[blockIdx.x * n + i]; 
      out[blockIdx.x * n + i] = acc; 
    }
  }
}


__global__ void mid_kernel(int n, int *in, int *out) { 
  
  if (threadIdx.x == 0){ 
    int acc = 0; 
    
    for (int i = 0; i < n; ++i){ 
      acc += in[(i+1)*32 - 1]; 
      out[i] = acc; 
    }
  }
}


__global__ void post_kernel(int n, int *imm, int *aux, int *out) { 
  
 
  if (threadIdx.x == 0){
    
    int val = aux[blockIdx.x-1]; 
    if (blockIdx.x == 0) val = 0; 
    
    for (int i = 0; i < n; ++i){ 
      
      out[blockIdx.x * n + i] = imm[blockIdx.x * n + i] + val; 
    }
  }
} 




#define ELT_PER_BLOCK 32
#define NUM_BLOCKS    32

#define SIZE (ELT_PER_BLOCK * NUM_BLOCKS)
int main(int argc, char **argv) {
 
  int *in = NULL; 
  int *out = NULL; 
  
  // debug 
  int *imm = NULL; 
  int *aux = NULL; 
  
  in = (int*)malloc(SIZE*sizeof(int)); 
  out = (int*)malloc(SIZE*sizeof(int)); 

  // debug 
  imm = (int*)malloc(SIZE*sizeof(int)); 
  aux = (int*)malloc(NUM_BLOCKS*sizeof(int)); 

  int *din = NULL; 
  
  int *imm1 = NULL; 
  int *aux1 = NULL; 

  int *dout = NULL;
  

  // Allocate all GPU Storage
  hipMalloc((void**)&din,SIZE*sizeof(int)); 
  hipMalloc((void**)&imm1,SIZE*sizeof(int)); 
  hipMalloc((void**)&aux1,NUM_BLOCKS*sizeof(int));
  hipMalloc((void**)&dout,SIZE*sizeof(int)); 

  // Generate some data 
  for (int i = 0; i < SIZE; i ++) { 
    in[i] = i+1;
  }
  
  
  hipMemcpy(din,in,SIZE*sizeof(int),hipMemcpyHostToDevice);

  // silly_kernel<<<1,1,0>>>(SIZE,din,dout); 

  silly_kernel<<<NUM_BLOCKS,ELT_PER_BLOCK,0>>>(ELT_PER_BLOCK,din,imm1);
  mid_kernel<<<1,NUM_BLOCKS,0>>>(ELT_PER_BLOCK,imm1,aux1); 
  post_kernel<<<NUM_BLOCKS,ELT_PER_BLOCK,0>>>(ELT_PER_BLOCK,imm1,aux1,dout); 
  
  hipMemcpy(out,dout, SIZE*sizeof(int),hipMemcpyDeviceToHost); 

  //debug 
  hipMemcpy(imm,imm1, SIZE*sizeof(int),hipMemcpyDeviceToHost); 
  hipMemcpy(aux,aux1, NUM_BLOCKS*sizeof(int),hipMemcpyDeviceToHost); 

  hipFree(din);
  hipFree(dout);
  hipFree(aux1);
  hipFree(imm1);
  
  for (int i = 0; i < SIZE; ++i) {
    printf("%d ", out[i]);
  }
  printf("\n---------------------------------------------------------------------------\n"); 


  for (int i = 0; i < NUM_BLOCKS; ++i) {
    printf("%d ", aux[i]);
  }
  printf("\n---------------------------------------------------------------------------\n"); 

  for (int i = 0; i < SIZE; ++i) {
    printf("%d ", imm[i]);
  }
  printf("\n---------------------------------------------------------------------------\n"); 

  
  

  return 0;
} 
